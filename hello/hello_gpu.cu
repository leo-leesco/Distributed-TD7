#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* Function computing the final string to print */
__global__ void compute_string(char *res, char *a, char *b, char *c,
                               int length) {
  int i;

  i = blockIdx.x * blockDim.x + threadIdx.x;

  id(i < N) { res[i] = a[i] + b[i] + c[i]; }
}

int main() {

  char *res;

  char a[30] = {40, 70, 70, 70, 80, 0, 50, 80, 80, 70, 70, 0,  40, 80, 79,
                70, 0,  40, 50, 50, 0, 70, 80, 0,  30, 50, 30, 30, 0,  0};
  char b[30] = {10, 10, 10, 10, 10, 10, 10, 10, 10, 10, 10, 10, 10, 10, 10,
                10, 10, 10, 10, 10, 10, 10, 10, 10, 10, 10, 10, 10, 0,  0};
  char c[30] = {22, 21, 28, 28, 21, 22, 27, 21, 24, 28, 20, 22, 20, 24, 22,
                29, 22, 21, 20, 25, 22, 25, 20, 22, 27, 25, 28, 25, 0,  0};

  res = (char *)malloc(30 * sizeof(char));

  /* This function call should be programmed in CUDA */
  /* -> need to allocate and transfer data to/from the device */

  char *da, *db, *dc, *dres;
  hipMalloc(&da, 30);
  hipMalloc(&db, 30);
  hipMalloc(&dc, 30);
  hipMalloc(&dres, 30);

  hipMemcpy(da, a, 30, hipMemcpyHostToDevice);
  hipMemcpy(db, b, 30, hipMemcpyHostToDevice);
  hipMemcpy(dc, c, 30, hipMemcpyHostToDevice);

  int nb_thr = 6;
  int nb_blk = 5;

  compute_string<<<nb_blk, nb_thr>>>(dres, da, db, dc, 30);

  hipMemcpy(res, dres, 30, hipMemcpyHostToDevice);
  printf("%s\n", res);

  hipFree(da);
  hipFree(db);
  hipFree(dc);
  hipFree(dres);

  free(res);

  return 0;
}
